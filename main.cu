#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <memory>

#define NUM_THREADS 10000
#define SIZE 10
#define BLOCK_WIDTH 100

__global__ void gpu_increment_atomic(int* d_a) {
	//�����̵߳�����
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//��10��Ԫ����ÿ���߳�����
	tid = tid % SIZE;
	//d_a[tid] += 1;
	atomicAdd(&d_a[tid], 1);    
	/*atomicAddԭ�Ӳ��������滻��֮ǰ��ֱ��+=�������ú�������2����������һ������������Ҫ����ԭ�Ӽӷ�����
	���ڴ����򣻵ڶ��������Ǹ�ԭ�Ӽӷ���������Ҫ���ϵ�ֵ��
	�ú�������߼��ϱ�֤��ÿ�����������̶߳���ͬ���ڴ������ϵġ���ȡ��ֵ-�ۼ�-��д��ֵ�������ǲ��ɱ�
	�����߳����ҵ�ԭ���Ե�������ɵġ�*/
}

int main(void) {
	printf("%d total threads in %d blocks writing into %d array elements\n", NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, SIZE);

	//�����ͷ��������ڴ�
	int h_a[SIZE];
	const int ARRAY_BYTES = SIZE * sizeof(int);

	//�����ͷ���GPU�ڴ�
	int* d_a;
	hipMalloc((void**)&d_a, ARRAY_BYTES);
	//��ʼ��GPU�ڴ棬Ĭ��ֵΪ0
	hipMemset((void*)d_a, 0, ARRAY_BYTES);
	gpu_increment_atomic << <NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >> > (d_a);

	//��GPU���ƻ��������Ҵ�ӡ����
	hipMemcpy(h_a, d_a, ARRAY_BYTES, hipMemcpyDeviceToHost);
	printf("Number of times a particular Array index has been incremented is:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("index:%d --> %d times\n", i, h_a[i]);
	}
	hipFree(d_a);
	return 0;
}

